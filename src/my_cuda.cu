#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADPTR dev_table = NULL;
RESULTPTR dev_res = NULL;
int *dev_LUT = NULL;

__global__ void logic_simulation_kernel(THREADPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid].output = output;
	}
}


extern "C" void dummy_gpu(int level){
	//int i;
	int blocks;

	//size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[level];

	//device_allocations();

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, cuda_tables[level], length*sizeof(THREADTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);


	blocks = (length+127)/128;
    logic_simulation_kernel<<<blocks,128>>>(dev_table,dev_res,length);


	HANDLE_ERROR( hipMemcpy(result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",result_tables[level][i]);
    //printf("\n");

    // Free device global memory
    //HANDLE_ERROR( hipFree(dev_table));
    //HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());
}



extern "C" void device_allocations()
{
	size_t size = patterns*maxgates;

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
}
