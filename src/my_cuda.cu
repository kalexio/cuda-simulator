#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;


__global__ void logic_simulation_kernel(THREADPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1D(texLUT,index);
		dev_res[tid].output = output;
	}
}


extern "C" void dummy_gpu(){
	//int i;
	int blocks;
	THREADPTR dev_table = NULL;
	RESULTPTR dev_res = NULL;
	int *dev_LUT = NULL;
	size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[0];
	hipError_t err = hipSuccess;


	//hipDeviceProp_t prop;

	//if(prop.deviceOverlap)
		//printf(" speed up from streams :)\n");

	// texture memory
	err = hipMalloc( (void**)&dev_LUT, 182*sizeof(int));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device memory for LUT (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	printf("LUT[0] = %d\n",LUT[0]);
	printf("Copy input data from the host memory LUT to the CUDA device dev_LUT\n");
	err = hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	err = hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int));
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to cudabind (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

    // cuda table
	err = hipMalloc( (void**)&dev_table, size);
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(dev_table, cuda_tables[0], size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	//res table
	err = hipMalloc( (void**)&dev_res, patterns*levels[0]*sizeof(int));
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

	printf("%d\n",patterns*levels[0]);
	blocks = (patterns*levels[0]+127)/128;
    logic_simulation_kernel<<<blocks,128>>>(dev_table,dev_res,length);

	printf("Copy input data from the device memory to the host\n");
	err = hipMemcpy(result_tables[0], dev_res,patterns*levels[0]*sizeof(int) , hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to copy the results from device to host (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}


    printf("END\n");

    //for (i = 0; i<54; i++ )
    	//printf("%d",result_tables[0][i]);

    // Free device global memory
    err = hipFree(dev_table);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device table (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(dev_res);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device result (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


}
