#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" {
#include "my_cuda.h"
}
#include <stdio.h>

__global__ void dummy_gpu_kernel(int a, int b, int *c){
	*c = a + b;
}


extern "C" void dummy_gpu(){
   int c;
   int *dev_c;
   hipMalloc( (void**)&dev_c, sizeof(int));
   dummy_gpu_kernel<<<1,1>>>(2,7,dev_c);
   hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
   printf("2 + 7 = %d\n",c);
   printf("executed!\n");
   hipFree(dev_c);
}
