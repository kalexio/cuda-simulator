#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADFAULTPTR dev_table = NULL;
THREADFAULTPTR dev_table2 = NULL;
//THREADFAULTPTR dev_table3 = NULL;
RESULTPTR dev_res = NULL;
RESULTPTR dev_res2 = NULL;
//RESULTPTR Goodsim = NULL;
int *dev_LUT = NULL;
int *cuda_vecs = NULL;
int Cuda_index = 0;
int real_faults = -1;
//int total=0;


__global__ void fill_struct_kernel(THREADFAULTPTR dev_table, int* Vectors, int offset, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[0] = Vectors[thread_id];
		dev_table[thread_id].m0 = 1;
		dev_table[thread_id].m1 = 0;
	}
}


__global__ void fill_struct_kernel1(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int read_mem, int k){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[tid+read_mem].output;
		dev_table[thread_id].m0 = 1;
		dev_table[thread_id].m1 = 0;
	}
}



__global__ void logic_simulation_kernel(THREADFAULTPTR dev_table, RESULTPTR dev_res, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//elegxos an perasoume ta osa steilame
	//xreiazomaste kai allon ena elegxo an ayta pou exoume steilei einai pio polla apo ta nhmata
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid+pos];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid+pos].output = output;
	}
}




__global__ void fill_fault_struct_kernel_PI(THREADFAULTPTR dev_table, int* Vectors, int offset, int length, int pos, int inj_bit0, int inj_bit1, int gatepos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[0] = Vectors[tid+gatepos];
		dev_table[thread_id].m0 = inj_bit0;
		dev_table[thread_id].m1 = inj_bit1;
	}
}

__global__ void fill_fault_struct_kernel_notPI(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int inj_bit0, int inj_bit1, int gatepos, int k){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[tid+gatepos].output;
		dev_table[thread_id].m0 = inj_bit0;
		dev_table[thread_id].m1 = inj_bit1;
	}
}


__global__ void fill_fault_struct_kernel_Paths(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int gatepos, int k){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid + gatepos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[tid+pos].output;
		dev_table[thread_id].m0 = 1;
		dev_table[thread_id].m1 = 0;
	}
}

__global__ void fault_injection_kernel(THREADFAULTPTR dev_table2, RESULTPTR dev_res2, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table2[tid+pos];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res2[tid+pos].output = (output & data.m0) | data.m1;
	}
}


__global__ void fault_detection_kernel(THREADFAULTPTR dev_table,RESULTPTR dev_res,RESULTPTR Good,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid];
		RESULTYPE data1 = Good[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		int output1 = data1.output;
		dev_res[tid].output = output ^ output1;
	}
}







extern "C" void device_allocations()
{

	size_t size = patterns*total_gates;
	//int size = 1000000;

	//int dev;
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
	//allocations for cuda_vectors
	HANDLE_ERROR( hipMalloc( (void**)&cuda_vecs, (patterns*levels[0]) * sizeof(int) ));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADFAULTYPE)));
	// check the cuda mem set with a memcpy ok -----------------------------------------------
	HANDLE_ERROR(hipMemset(dev_table, 0, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));


	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
	//fill the Cuda_vecs
	HANDLE_ERROR( hipMemcpy(cuda_vecs, cuda_vectors, (patterns*levels[0]) * sizeof(int) , hipMemcpyHostToDevice));
}



extern "C" void init_first_level()
{
	int offset, i, threads, blocks;
	int length, pos;

	threads = 128;
	blocks = ( patterns + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( patterns + (threads-1))/threads;
	}

	offset = PI;
	for (i = 0; i<levels[0]; i++) {
		pos = i * patterns;
		fill_struct_kernel<<<blocks,threads>>>(dev_table, cuda_vecs, offset, patterns, pos);
	}
	threads = 128;
	length = patterns * levels[0];
	blocks = ( length + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( length + (threads-1))/threads;
	}
	logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
	Cuda_index = length;
}



extern "C" void init_any_level()
{
	int i, j, k, offset, array;
	int threads, blocks, pos1, length;
	int pos = 0;
	GATEPTR cg, hg;

	pos = levels[0]-1;

	//Gia ola ta epipeda tou kyklwmatos
	//mhpws thelei -2 anti gia -1????
	for (i = 1; i< (maxlevel-1); i++){

		threads = 128;
		blocks = ( patterns + (threads-1))/threads;
		if (blocks < 200) {
			threads = 64;
			blocks = ( patterns + (threads-1))/threads;
		}

		for (j = 0; j< levels[i]; j++){
			cg = event_list[i].list[j];
			offset = find_offset(cg);
			pos++;
			pos1 = pos*patterns;
			//printf("i am %s with %d offset\n", cg->symbol->symbol,offset);
			for (k = 0; k<cg->ninput; k++) {
				hg = cg->inlis[k];
				array = hg->index * patterns;
				fill_struct_kernel1<<<blocks,threads>>>(dev_table, dev_res, offset, patterns, pos1, array, k);
			}
		}

		threads = 128;
		length = patterns * levels[i];
		blocks = ( length + (threads-1))/threads;
		if (blocks < 200) {
			threads = 64;
			blocks = ( length + (threads-1))/threads;
		}

		logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
		Cuda_index = Cuda_index + length;

	}
	//HANDLE_ERROR( hipMemcpy(result_tables, dev_res, Cuda_index* sizeof(int) , hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipFree(dev_table));
}



extern "C" void device_allocations2()
{
	size_t size = 126000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res2, size*sizeof(int)));
}



extern "C" void fault_init_first_level(){
		int i, k;
		GATEPTR cg, hg;
		int inj_bit0 = 1;
		int inj_bit1 = 0;
		int blocks, threads, length;
		int  gatepos, array, arr;
		//int real_faults = -1;

		threads = 256;
		blocks = ( patterns + (threads-1))/threads;
		//if (blocks < 200) {
			//threads = 64;
			//blocks = ( patterns + (threads-1))/threads;
		//}

		for (i = 0; i<total_faults; i++){
			//vriskw to offset ths pulhs kai ta injection bits eite einai PI PO etc
			cg = fault_list[i].gate;
			cg->TFO_list[i] = 1;

			if (fault_list[i].SA == 0) {
				inj_bit0 = 0;
				inj_bit1 = 0;
			}
			else inj_bit1 = 1;

			if ( cg->outlis[0]->fn != PO ) {
				real_faults++;
				//thesh stou pinakes twn faults
				cg->fault_level[i] = 0;
				cg->flevel_pos[i] = real_faults;

				if (cg->fn != PI) {
					for (k = 0; k<cg->ninput; k++) {
						hg = cg->inlis[k];
						array = hg->index * patterns;
						arr = real_faults*patterns;
						//printf("%s me index %d  ",hg->symbol->symbol,hg->index);

						fill_fault_struct_kernel_notPI<<<blocks,threads>>>(dev_table2, dev_res, cg->offset, patterns, arr, inj_bit0, inj_bit1, array, k);

					}
				}

				else {
					//Einai PI
					gatepos = cg->level_pos*patterns;
					//printf("%s %d ",cg->symbol->symbol,cg->level_pos);
					arr = real_faults*patterns;

					fill_fault_struct_kernel_PI<<<blocks,threads>>>(dev_table2, cuda_vecs, PI, patterns, arr, inj_bit0, inj_bit1, gatepos);
			   }

			}

			else {
				fault_list[i].end = 2;
			}//end of else

		}//end for faults

		//Call fault injection

		Cuda_index = 0;
		length = real_faults*patterns;
		printf("lenth in first level %d\n",length);
		threads = 512;
		blocks = ( length + (threads-1))/threads;
		fault_injection_kernel<<<blocks,threads>>>(dev_table2, dev_res2, length, Cuda_index);
		Cuda_index = length;
}


extern "C" void fault_init_any_level(){
	int i, k;
	GATEPTR cg, hg;
	int  array, arr;
	int threads, blocks, length;
	//int real_faults = -1;
	int counter = -1;

	threads = 256;
	blocks = ( patterns + (threads-1))/threads;

	for (i = 0; i<total_faults; i++){
		if (fault_list[i].end != 1) {
			if(fault_list[i].TFO_stack.list[fault_list[i].TFO_stack.last]->outlis[0]->fn == PO){
				fault_list[i].end = 2;
			}
			//not PO yet
			else{
				while (fault_list[i].affected_gates > 0){
					fault_list[i].affected_gates--;
					cg = fault_list[i].TFO_stack.list[(fault_list[i].TFO_stack.last)--];
					real_faults++;
					counter++;

					//cg->fault_level[i] = loop;
					cg->flevel_pos[i] = real_faults;

					for (k = 0; k<cg->ninput; k++){
						hg = cg->inlis[k];

						//den einai sto path ara diavase apo to non fault table
						if (hg->TFO_list[i] != 1){
							//from where it reads
							array = hg->index*patterns;
							//to where it will write
							arr = real_faults*patterns;

							//CALL KERNEL
							fill_fault_struct_kernel_Paths<<<blocks,threads>>>(dev_table2, dev_res, cg->offset,patterns,array,arr,k);

						}//end of if path
						else{
							//from where it reads
							array = hg->flevel_pos[i]*patterns;
							//to where it will write
							arr = real_faults*patterns;

							//CALL KERNEL
							fill_fault_struct_kernel_Paths<<<blocks,threads>>>(dev_table2, dev_res2, cg->offset,patterns,array,arr,k);

						}//end of else path

					}//end of inputs

				}//end of while

			}//end of else

		}//end of fault_list.end != 1
	}//end of total faults

	length = counter*patterns;
	threads = 512;
	blocks = ( length + (threads-1))/threads;
	fault_injection_kernel<<<blocks,threads>>>(dev_table2, dev_res2, length, Cuda_index);
	Cuda_index = Cuda_index + length;
	printf("lenth %d\n",Cuda_index);

}


extern "C" void device_deallocations2()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table2));
    HANDLE_ERROR( hipFree(dev_LUT));
    HANDLE_ERROR( hipFree(cuda_vecs));
    HANDLE_ERROR( hipFree(dev_res2));
    HANDLE_ERROR( hipFree(dev_res));
    HANDLE_ERROR( hipDeviceReset());

}


extern "C" int find_offset (GATEPTR cg)
{
	int inputs, offset, fn;

	inputs = cg->ninput;
	fn = cg->fn;

	switch (fn) {
		case PI: offset = 0;
			break;
		case AND:
			if (inputs == 2) offset = AND2;
			else if (inputs == 3) offset = AND3;
			else if (inputs == 4) offset = AND4;
			else if (inputs == 5) offset = AND5;
			break;
		case NAND:
			if (inputs == 2) offset = NAND2;
			else if (inputs == 3) offset = NAND3;
			else if (inputs == 4) offset = NAND4;
			break;
		case OR:
			if (inputs == 2) offset = OR2;
			else if (inputs == 3) offset = OR3;
			else if (inputs == 4) offset = OR4;
			else if (inputs == 5) offset = OR5;
			break;
		case NOR:
			if (inputs == 2) offset = NOR2;
			else if (inputs == 3) offset = NOR3;
			else if (inputs == 4) offset = NOR4;
			break;
		case PO: offset = PO;
	}

	return (offset);

}



/*
extern "C" void dummy_gpu(int level){
	//int i;
	int blocks;
	int threads;


	//size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[level];
	//total=total+length;
	//printf("Length for logic sim epipedo %d %d\n",level,length);

	//device_allocations();

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, cuda_tables[level], length*sizeof(THREADTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
    logic_simulation_kernel<<<blocks,threads>>>(dev_table,dev_res,length);


	HANDLE_ERROR( hipMemcpy(result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",result_tables[level][i]);
    //printf("\n");
}


extern "C" void dummy_gpu2(int level){
	//int i;
	int blocks;
	int threads;

	int length = no_po_faults*patterns;
	if (level > 0) length = next_level_length * patterns;
	//total=total+length;
	//printf("CUDA2 length gia epipedo %d %d\n",level,length);

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table2, fault_tables[level], length*sizeof(THREADFAULTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
	fault_injection_kernel<<<blocks,threads>>>(dev_table2,dev_res,length);
	

	HANDLE_ERROR( hipMemcpy(fault_result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",fault_result_tables[level][i]);
}



extern "C" void dummy_gpu3(){
	//int i;
	int blocks;
	int threads;

	int length = detect_index*patterns;
	//printf("CUDA3 length is %d\n",length);

	//printf("I am here\n");
	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table3, detect_tables, length*sizeof(THREADFAULTYPE), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(Goodsim, GoodSim, length*sizeof(int), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
	fault_detection_kernel<<<blocks,threads>>>(dev_table3,dev_res,Goodsim,length);


	HANDLE_ERROR( hipMemcpy(Final, dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));

    //for(i = 0; i<length; i++)printf("%d",GoodSim[i]);

    printf("length is %d\n",length);

    //for (i = 0; i<length; i++ )
    	//printf("%d",Final[i]);
}



extern "C" void device_allocations()
{
	size_t size = patterns*maxgates;
	//int dev;

	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);
	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
}


extern "C" void device_allocations2()
{
	size_t size = 100000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
}


extern "C" void alloc(){
	size_t size = 100000000;
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADTYPE)));
}


extern "C" void device_allocations3()
{
	int length = detect_index*patterns;
	//int length =100000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table3, length*sizeof(THREADFAULTYPE)));
	HANDLE_ERROR( hipMalloc( (void**)&Goodsim, length*sizeof(int)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, length*sizeof(int)));
}


extern "C" void device_deallocations()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());

}


extern "C" void dealloc(){
	HANDLE_ERROR( hipFree(dev_table2));
}


extern "C" void device_deallocations2()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table2));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());
}

extern "C" void device_deallocations3()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table3));
    HANDLE_ERROR( hipFree(dev_res));
    HANDLE_ERROR( hipDeviceReset());
}
*/
