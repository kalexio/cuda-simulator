#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADFAULTPTR dev_table = NULL;
THREADFAULTPTR dev_table2 = NULL;
THREADFAULTPTR dev_table3 = NULL;
RESULTPTR dev_res = NULL;
RESULTPTR dev_res2 = NULL;
RESULTPTR Goodsim = NULL;
RESULTPTR Detection_table = NULL;
int *dev_LUT = NULL;
int *cuda_vecs = NULL;
int Cuda_index = 0;
int real_faults = -1;
int *patterns_positions;
//int total=0;


__global__ void fill_struct_kernel(THREADFAULTPTR dev_table, int* Vectors, int offset, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[0] = Vectors[thread_id];
		dev_table[thread_id].m0 = 1;
		//dev_table[thread_id].m1 = 0;
	}
}


__global__ void fill_struct_kernel1(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int read_mem, int k){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[tid+read_mem].output;
		dev_table[thread_id].m0 = 1;
		//dev_table[thread_id].m1 = 0;
	}
}



__global__ void logic_simulation_kernel(THREADFAULTPTR dev_table, RESULTPTR dev_res, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//elegxos an perasoume ta osa steilame
	//xreiazomaste kai allon ena elegxo an ayta pou exoume steilei einai pio polla apo ta nhmata
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid+pos];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid+pos].output = output;
	}
}




__global__ void fill_fault_struct_kernel_PI(THREADFAULTPTR dev_table, int* Vectors, int offset, int length, int pos, int inj_bit0, int inj_bit1, int gatepos,int *patterns_positions){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		int index = patterns_positions[thread_id];
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[0] = Vectors[index];
		dev_table[thread_id].m0 = inj_bit0;
		dev_table[thread_id].m1 = inj_bit1;
	}
}

__global__ void fill_fault_struct_kernel_notPI(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int inj_bit0, int inj_bit1, int gatepos, int k, int *patterns_positions){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid+pos;
		int index = patterns_positions[thread_id];
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[index+gatepos].output;
		dev_table[thread_id].m0 = inj_bit0;
		dev_table[thread_id].m1 = inj_bit1;
	}
}

__global__ void fill_fault_struct_kernel_Paths(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int gatepos, int k, int *patterns_positions,int until_now){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid + gatepos;
		int index = patterns_positions[tid + until_now];
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[index+pos].output;
		//exei ginei memset
		dev_table[thread_id].m0 = 1;
		//dev_table[thread_id].m1 = 0;
	}
}

__global__ void fault_injection_kernel(THREADFAULTPTR dev_table2, RESULTPTR dev_res2, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table2[tid+pos];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res2[tid+pos].output = (output & data.m0) | data.m1;
	}
}


__global__ void fill_detetection_struct(THREADFAULTPTR dev_table, RESULTPTR dev_res,int offset, int length, int pos, int gatepos,int k, int *patterns_positions,int until_now,int good_index,RESULTPTR Goodsim,RESULTPTR devres){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		int thread_id = tid + gatepos;
		int index = patterns_positions[tid + until_now];
		dev_table[thread_id].offset = offset;
		dev_table[thread_id].input[k] = dev_res[index+pos].output;
		dev_table[thread_id].m0 = 1;
		Goodsim[thread_id] = devres[good_index+index];
		//dev_table[thread_id].m1 = 0;
	}
}


__global__ void fault_detection_kernel(THREADFAULTPTR dev_table,RESULTPTR dev_res,RESULTPTR Good,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid];
		RESULTYPE data1 = Good[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		int output1 = data1.output;
		dev_res[tid].output = output ^ output1;
	}
}




extern "C" void device_allocations()
{

	size_t size = patterns*total_gates;
	//int size = 1000000;

	//int dev;
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	HANDLE_ERROR( hipSetDevice (0));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
	//allocations for cuda_vectors
	HANDLE_ERROR( hipMalloc( (void**)&cuda_vecs, (patterns*levels[0]) * sizeof(int) ));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADFAULTYPE)));
	// check the cuda mem set with a memcpy ok -----------------------------------------------
	HANDLE_ERROR(hipMemset(dev_table, 0, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));


	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
	//fill the Cuda_vecs
	HANDLE_ERROR( hipMemcpy(cuda_vecs, cuda_vectors, (patterns*levels[0]) * sizeof(int) , hipMemcpyHostToDevice));
}



extern "C" void init_first_level()
{
	int offset, i, threads, blocks;
	int length, pos;

	threads = 128;
	blocks = ( patterns + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( patterns + (threads-1))/threads;
	}

	offset = PI;
	for (i = 0; i<levels[0]; i++) {
		pos = i * patterns;
		fill_struct_kernel<<<blocks,threads>>>(dev_table, cuda_vecs, offset, patterns, pos);
	}
	threads = 128;
	length = patterns * levels[0];
	blocks = ( length + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( length + (threads-1))/threads;
	}
	logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
	Cuda_index = length;
}



extern "C" void init_any_level()
{
	int i, j, k, offset, array;
	int threads, blocks, pos1, length;
	int pos = 0;
	GATEPTR cg, hg;

	pos = levels[0]-1;

	//Gia ola ta epipeda tou kyklwmatos
	//mhpws thelei -2 anti gia -1????
	for (i = 1; i< (maxlevel-1); i++){

		threads = 128;
		blocks = ( patterns + (threads-1))/threads;
		if (blocks < 200) {
			threads = 64;
			blocks = ( patterns + (threads-1))/threads;
		}

		for (j = 0; j< levels[i]; j++){
			cg = event_list[i].list[j];
			offset = find_offset(cg);
			pos++;
			pos1 = pos*patterns;
			//printf("i am %s with %d offset\n", cg->symbol->symbol,offset);
			for (k = 0; k<cg->ninput; k++) {
				hg = cg->inlis[k];
				array = hg->index * patterns;
				fill_struct_kernel1<<<blocks,threads>>>(dev_table, dev_res, offset, patterns, pos1, array, k);
			}
		}

		threads = 128;
		length = patterns * levels[i];
		blocks = ( length + (threads-1))/threads;
		if (blocks < 200) {
			threads = 64;
			blocks = ( length + (threads-1))/threads;
		}

		logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
		Cuda_index = Cuda_index + length;

	}
	//HANDLE_ERROR( hipMemcpy(result_tables, dev_res, Cuda_index* sizeof(int) , hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipFree(dev_table));
}



extern "C" void device_allocations2(int tot)
{
	size_t size = 120000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	//printf("CUDA tot = %d\n",tot);
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADFAULTYPE)));
	HANDLE_ERROR(hipMemset(dev_table2, 0, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res2, size*sizeof(int)));
	HANDLE_ERROR( hipMalloc( (void**)&patterns_positions, tot*sizeof(int)));
	HANDLE_ERROR( hipMemcpy(patterns_positions, patterns_posit, tot * sizeof(int) , hipMemcpyHostToDevice));
}



extern "C" void fault_init_first_level(){
		int i, k;
		GATEPTR cg, hg;
		int inj_bit0 = 1;
		int inj_bit1 = 0;
		int blocks, threads, length;
		int  gatepos, array, arr;
		//int real_faults = -1;

		threads = 256;
		blocks = ( patterns + (threads-1))/threads;
		//if (blocks < 200) {
			//threads = 64;
			//blocks = ( patterns + (threads-1))/threads;
		//}

		for (i = 0; i<total_faults; i++){
			//vriskw to offset ths pulhs kai ta injection bits eite einai PI PO etc
			cg = fault_list[i].gate;
			cg->TFO_list[i] = 1;

			if (fault_list[i].SA == 0) {
				inj_bit0 = 0;
				inj_bit1 = 0;
			}
			else inj_bit1 = 1;

			if ( cg->outlis[0]->fn != PO ) {
				real_faults++;

				//thesh stou pinakes twn faults
				//allagh = until_now??
				//cg->flevel_pos[i] = real_faults;
				cg->flevel_pos[i] = fault_list[i].until_now;

				if (cg->fn != PI) {
					for (k = 0; k<cg->ninput; k++) {
						hg = cg->inlis[k];
						//apo pou tha ksekinhsei na diavazei (Logis sim pinaka)
						array = hg->index * patterns;
						//pou tha grapsei
						//arr = real_faults*patterns;
						arr = fault_list[i].until_now;
						//printf("%s me index %d  ",hg->symbol->symbol,hg->index);

						fill_fault_struct_kernel_notPI<<<blocks,threads>>>(dev_table2, dev_res, cg->offset, fault_list[i].tot_patterns, arr, inj_bit0, inj_bit1, array, k,patterns_positions);

					}
				}

				else {
					//Einai PI
					//apo pou tha diavsei
					gatepos = cg->level_pos*patterns;
					//printf("%s %d ",cg->symbol->symbol,cg->level_pos);
					//pou tha grapsei
					//arr = real_faults*patterns;
					arr = fault_list[i].until_now;

					fill_fault_struct_kernel_PI<<<blocks,threads>>>(dev_table2, cuda_vecs, PI, fault_list[i].tot_patterns, arr, inj_bit0, inj_bit1, gatepos,patterns_positions);
			   }

			}

			else {
				fault_list[i].end = 2;
			}//end of else

		}//end for faults

		//Call fault injection

		Cuda_index = 0;
		//length = real_faults*patterns;
		length = tot_patterns;
		printf("lenth in first level %d\n",length);
		threads = 512;
		blocks = ( length + (threads-1))/threads;
		fault_injection_kernel<<<blocks,threads>>>(dev_table2, dev_res2, length, Cuda_index);
		Cuda_index = length;
}


extern "C" int fault_init_any_level(){
	int i, k;
	GATEPTR cg, hg;
	int  array, arr;
	int threads, blocks, length;
	int counter = -1;
	int prev_tot_patterns;

	prev_tot_patterns = tot_patterns;

	for (i = 0; i<total_faults; i++){
		if (fault_list[i].end != 1) {
			if(fault_list[i].TFO_stack.list[fault_list[i].TFO_stack.last]->outlis[0]->fn == PO){
				fault_list[i].end = 1;
			}
			//not PO yet
			else{
				while (fault_list[i].affected_gates > 0){
					fault_list[i].affected_gates--;
					cg = fault_list[i].TFO_stack.list[(fault_list[i].TFO_stack.last)--];
					real_faults++;
					counter++;

					//allagh = tot_patterns;??
					//cg->flevel_pos[i] = real_faults;
					cg->flevel_pos[i] = tot_patterns;

					for (k = 0; k<cg->ninput; k++){
						hg = cg->inlis[k];

						//den einai sto path ara diavase apo to non fault table
						if (hg->TFO_list[i] != 1){
							//from where it reads
							array = hg->index*patterns;
							//to where it will write
							//arr = tot_patterns kai sto telos tou kernel tot_pattern = tot_pattern + falt_list.tot;
							//mallon tha auksanetai exw apo thn if auth kai tha einai kai gia tis 2 periptwseis

							//arr = real_faults*patterns;
							arr = tot_patterns;

							//CALL KERNEL
							threads = 256;
							blocks = ( fault_list[i].tot_patterns + (threads-1))/threads;
							fill_fault_struct_kernel_Paths<<<blocks,threads>>>(dev_table2, dev_res, cg->offset,fault_list[i].tot_patterns,array,arr,k,patterns_positions,fault_list[i].until_now);

						}//end of if path
						else{
							//from where it reads
							//!!!!!idea na xrhsimopoihsoume th metablhth flevel pos gia na mas deixnei th thesh tou pinka
							//array = hg->flevel_pos[i];

							//array = hg->flevel_pos[i]*patterns;
							array = hg->flevel_pos[i];

							//to where it will write
							//!!!!allagh opws eipame

							//arr = real_faults*patterns;
							arr = tot_patterns;

							//CALL KERNEL
							threads = 256;
							blocks = ( fault_list[i].tot_patterns + (threads-1))/threads;
							fill_fault_struct_kernel_Paths<<<blocks,threads>>>(dev_table2, dev_res2, cg->offset,fault_list[i].tot_patterns,array,arr,k,patterns_positions,fault_list[i].until_now);

						}//end of else path

					}//end of inputs

					//mallon edw paei h auxhsh tou tot_patterns!!!!!!!!!
					tot_patterns = tot_patterns + fault_list[i].tot_patterns;

				}//end of while

			}//end of else

		}//end of fault_list.end != 1
	}//end of total faults

	//length = counter*patterns;
	length = tot_patterns - prev_tot_patterns;
	threads = 512;
	blocks = ( length + (threads-1))/threads;
	fault_injection_kernel<<<blocks,threads>>>(dev_table2, dev_res2, length, Cuda_index);
	Cuda_index = Cuda_index + length;
	printf("lenth until now %d\n",Cuda_index);

	return length;
}


extern "C" void device_allocations3(int tot)
{
	HANDLE_ERROR( hipFree(dev_table2));
	HANDLE_ERROR( hipFree(cuda_vecs));
	HANDLE_ERROR( hipMalloc( (void**)&dev_table3, (tot)*sizeof(THREADFAULTYPE)));
	HANDLE_ERROR( hipMalloc( (void**)&Goodsim, (tot)*sizeof(int)));
	HANDLE_ERROR( hipMalloc( (void**)&Detection_table, (tot)*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_table3, 0, (tot)*sizeof(THREADFAULTYPE)));
}


//prepei na eisagw sto fill kernels kai tis times tou goodsim
extern "C" void prepare_detection_table()
{
	int i, k;
	GATEPTR cg, hg;
	int counter = 0;
	int array, arr;
	int threads,blocks;
	int good_index;

	printf("i am in detect\n");

	for (i = 0; i<total_faults; i++){
		if ((fault_list[i].end == 1) ||(fault_list[i].TFO_stack.list[fault_list[i].TFO_stack.last]->outlis[0]->fn == PO)) {
			//printf("Arxiko sfalma %s\n",fault_list[i].gate->symbol->symbol);
			//printf("Exoume %d pules mesa sto %d sfalma\n",fault_list[i].TFO_stack.last,i);
			while(fault_list[i].TFO_stack.last>=0){

				//an exei mono ena sfalma tote prpeei na ginei injection
				cg = fault_list[i].TFO_stack.list[(fault_list[i].TFO_stack.last)--];

				//apo pou tha diabasei gia to kalo kuklwma gia to detection
				good_index = cg->index*patterns;

				//printf("lista %s\n",cg->symbol->symbol);
				//offset = find_offset(cg);

				//pare ta apotelesmata kai valta ston pinaka detect
				for (k = 0; k<cg->ninput; k++) {
					hg = cg->inlis[k];
					//printf("read inputs %s\n",hg->symbol->symbol);
					if(hg->TFO_list[i] != 1){

						//h thesh ston apothkeumeno pinaka
						array = hg->index*patterns;
						//h thesh ston pinaka pou ftiaxnoume prepei na auxhthei meta
						arr = counter;
						threads = 256;
						blocks = ( fault_list[i].tot_patterns + (threads-1))/threads;
						fill_detetection_struct<<<threads,blocks>>>(dev_table3, dev_res, cg->offset,fault_list[i].tot_patterns,array,arr,k,patterns_positions,fault_list[i].until_now,good_index,Goodsim,dev_res);
					}
					else{
						//h thesh ston apothkeumeno pinaka
						array = hg->flevel_pos[i];
						//h thesh ston pinaka pou ftiaxnoume
						arr = counter;
						threads = 256;
						blocks = ( fault_list[i].tot_patterns + (threads-1))/threads;
						fill_detetection_struct<<<threads,blocks>>>(dev_table3, dev_res2, cg->offset,fault_list[i].tot_patterns,array,arr,k,patterns_positions,fault_list[i].until_now,good_index,Goodsim,dev_res);
					}
				}//end for inputs
				counter = counter + fault_list[i].tot_patterns;
			}//end of while

		}//if checking
		else printf("something went wrong!\n");
	}//end of faults

	//CALL fault detection kernel
	printf("Synoliko %d",counter);
	fault_detection_kernel<<<threads,blocks>>>(dev_table3,Detection_table,Goodsim, counter);
}







extern "C" void device_deallocations3()
{
    // Free device global memory
	HANDLE_ERROR( hipFree(dev_table3));
	HANDLE_ERROR( hipFree(Goodsim));
    HANDLE_ERROR( hipFree(dev_LUT));
    HANDLE_ERROR( hipFree(dev_res2));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());
}


extern "C" int find_offset (GATEPTR cg)
{
	int inputs, offset, fn;

	inputs = cg->ninput;
	fn = cg->fn;

	switch (fn) {
		case PI: offset = 0;
			break;
		case AND:
			if (inputs == 2) offset = AND2;
			else if (inputs == 3) offset = AND3;
			else if (inputs == 4) offset = AND4;
			else if (inputs == 5) offset = AND5;
			break;
		case NAND:
			if (inputs == 2) offset = NAND2;
			else if (inputs == 3) offset = NAND3;
			else if (inputs == 4) offset = NAND4;
			break;
		case OR:
			if (inputs == 2) offset = OR2;
			else if (inputs == 3) offset = OR3;
			else if (inputs == 4) offset = OR4;
			else if (inputs == 5) offset = OR5;
			break;
		case NOR:
			if (inputs == 2) offset = NOR2;
			else if (inputs == 3) offset = NOR3;
			else if (inputs == 4) offset = NOR4;
			break;
		case PO: offset = PO;
	}

	return (offset);

}
