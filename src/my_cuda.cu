#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADFAULTPTR dev_table = NULL;
//THREADFAULTPTR dev_table2 = NULL;
//THREADFAULTPTR dev_table3 = NULL;
RESULTPTR dev_res = NULL;
//RESULTPTR Goodsim = NULL;
int *dev_LUT = NULL;
int *cuda_vecs = NULL;
int Cuda_index = 0;
//int total=0;


__global__ void fill_struct_kernel(THREADFAULTPTR dev_table, int* Vectors, int offset, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		//THREADTYPE data = dev_table[tid];
		int loc_pos = pos*length;
		dev_table[tid+loc_pos].offset = offset;
		dev_table[tid+loc_pos].input[0] = Vectors[tid];
		//an douleyei to memset tote poulo ayta
		//dev_table[tid].input[1] = 0;
		//dev_table[tid].input[2] = 0;
		//dev_table[tid].input[3] = 0;
		dev_table[tid+loc_pos].m0 = 1;
		dev_table[tid+loc_pos].m1 = 0;
	}
}


__global__ void fill_struct_kernel1(THREADFAULTPTR dev_table, RESULTPTR dev_res, int offset, int length, int pos, int read_mem, int k){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		//THREADTYPE data = dev_table[tid];
		int loc_pos = pos*length;
		dev_table[tid+loc_pos].offset = offset;
		dev_table[tid+loc_pos].input[k] = dev_res[read_mem].output;
		dev_table[tid+loc_pos].m0 = 1;
		dev_table[tid+loc_pos].m1 = 0;
	}
}



__global__ void logic_simulation_kernel(THREADFAULTPTR dev_table, RESULTPTR dev_res, int length, int pos){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//elegxos an perasoume ta osa steilame
	//xreiazomaste kai allon ena elegxo an ayta pou exoume steilei einai pio polla apo ta nhmata
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid+pos];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid+pos].output = output;
	}
}


__global__ void fault_injection_kernel(THREADFAULTPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid].output = (output && data.m0) || data.m1;
	}
}


__global__ void fault_detection_kernel(THREADFAULTPTR dev_table,RESULTPTR dev_res,RESULTPTR Good,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid];
		RESULTYPE data1 = Good[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		int output1 = data1.output;
		dev_res[tid].output = output ^ output1;
	}
}







extern "C" void device_allocations()
{
	//allocate memory for all the gates for logic sim
	//isws prepei na megalwsei gia na mhn ksanadesmeuoume mnhmh gia to fault sim
	size_t size = patterns*nog;

	//int dev;
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
	//allocations for cuda_vectors
	HANDLE_ERROR( hipMalloc( (void**)&cuda_vecs, (patterns*levels[0]) * sizeof(int) ));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADFAULTYPE)));
	// check the cuda mem set with a memcpy ok -----------------------------------------------
	HANDLE_ERROR(hipMemset(dev_table, 0,size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));

	//HANDLE_ERROR( hipMemcpy(cuda_table, dev_table, size*sizeof(THREADFAULTYPE),hipMemcpyDeviceToHost ));

	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
	//fill the Cuda_vecs
	HANDLE_ERROR( hipMemcpy(cuda_vecs, cuda_vectors, (patterns*levels[0]) * sizeof(int) , hipMemcpyHostToDevice));
}



extern "C" void init_first_level()
{
	int offset, i, threads, blocks;
	int length = patterns * levels[0];
	//GATEPTR cg;

	threads = 128;
	blocks = ( patterns + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( patterns + (threads-1))/threads;
	}

	for (i = 0; i<levels[0]; i++) {
		//call kernel for each gate
		//cg = event_list[0].list[i];
		offset = PI;
		//printf("i am %s with %d offset\n", cg->symbol->symbol,offset);

		//fill the first level of the array
		fill_struct_kernel<<<blocks,threads>>>(dev_table, cuda_vecs, offset, patterns, i);
	}

	threads = 128;
	blocks = ( length + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( length + (threads-1))/threads;
	}

	//do the first logic sim
	logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
	Cuda_index = length;
}



extern "C" void init_any_level()
{
	int i, j, k, offset;
	int epipedo, gatepos, array;
	int threads, blocks;
	int pos = 0;
	int length;
	GATEPTR cg, hg;

	pos = levels[0];

	//Gia ola ta epipeda tou kyklwmatos
	for (i = 1; i< (maxlevel-1); i++){



	length = patterns * levels[i];

	threads = 128;
	blocks = ( patterns + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( patterns + (threads-1))/threads;
	}

	for (j = 0; j< levels[i]; j++){
		cg = event_list[i].list[j];
		offset = find_offset(cg);
		//gia na to dwsoyme ston kernel fill
		pos++;
		//printf("i am %s with %d offset\n", cg->symbol->symbol,offset);
		for (k = 0; k<cg->ninput; k++) {
			hg = cg->inlis[k];
			array = hg->index * patterns;

			fill_struct_kernel1<<<blocks,threads>>>(dev_table, dev_res, offset, patterns, pos, array, k);
		}
	}

	threads = 128;
	blocks = ( length + (threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = ( length + (threads-1))/threads;
	}

	//do the first logic sim
	logic_simulation_kernel<<<blocks,threads>>>(dev_table, dev_res, length, Cuda_index);
	Cuda_index = Cuda_index + length;

	}

	HANDLE_ERROR( hipMemcpy(result_tables, dev_res, patterns*nog* sizeof(int) , hipMemcpyDeviceToHost));
}





/*void init_any_level(int lev,THREADPTR table)
{
	GATEPTR cg,hg,pg;
	int i,j,k,l,gatepos,m;
	register int pos;
	int epipedo;
	int offset,array,arr;

	//for all the gates of the lev level
	for (i = 0; i<=event_list[lev].last; i++) {
		cg = event_list[lev].list[i];

		//offset = find_offset(cg);
		//printf("%s\n",cg->symbol->symbol);
		//koita tis inlist kai pare th thesh twn pulwn apo tis opoies tha diavasoume

		for (k = 0; k<cg->ninput; k++) {
			hg = cg->inlis[k];
			epipedo = hg->level;
			gatepos = hg->level_pos;

			//opts
			array=gatepos*patterns;
			arr = i*patterns;

			//for all the patterns for this gate
			for ( j = 0; j<patterns; j++) {
				pos = arr + j;
				table[pos].offset = cg->offset;
				table[pos].input[k] = result_tables[epipedo][array+j].output;
			}
		}
	}
}*/




extern "C" int find_offset (GATEPTR cg)
{
	int inputs, offset, fn;

	inputs = cg->ninput;
	fn = cg->fn;

	switch (fn) {
		case AND:
			if (inputs == 2) offset = AND2;
			else if (inputs == 3) offset = AND3;
			else if (inputs == 4) offset = AND4;
			else if (inputs == 5) offset = AND5;
			break;
		case NAND:
			if (inputs == 2) offset = NAND2;
			else if (inputs == 3) offset = NAND3;
			else if (inputs == 4) offset = NAND4;
			break;
		case OR:
			if (inputs == 2) offset = OR2;
			else if (inputs == 3) offset = OR3;
			else if (inputs == 4) offset = OR4;
			else if (inputs == 5) offset = OR5;
			break;
		case NOR:
			if (inputs == 2) offset = NOR2;
			else if (inputs == 3) offset = NOR3;
			else if (inputs == 4) offset = NOR4;
			break;
		case PO: offset = PO;
	}

	return (offset);

}



/*
extern "C" void dummy_gpu(int level){
	//int i;
	int blocks;
	int threads;


	//size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[level];
	//total=total+length;
	//printf("Length for logic sim epipedo %d %d\n",level,length);

	//device_allocations();

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, cuda_tables[level], length*sizeof(THREADTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
    logic_simulation_kernel<<<blocks,threads>>>(dev_table,dev_res,length);


	HANDLE_ERROR( hipMemcpy(result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",result_tables[level][i]);
    //printf("\n");
}


extern "C" void dummy_gpu2(int level){
	//int i;
	int blocks;
	int threads;

	int length = no_po_faults*patterns;
	if (level > 0) length = next_level_length * patterns;
	//total=total+length;
	//printf("CUDA2 length gia epipedo %d %d\n",level,length);

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table2, fault_tables[level], length*sizeof(THREADFAULTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
	fault_injection_kernel<<<blocks,threads>>>(dev_table2,dev_res,length);
	

	HANDLE_ERROR( hipMemcpy(fault_result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",fault_result_tables[level][i]);
}



extern "C" void dummy_gpu3(){
	//int i;
	int blocks;
	int threads;

	int length = detect_index*patterns;
	//printf("CUDA3 length is %d\n",length);

	//printf("I am here\n");
	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table3, detect_tables, length*sizeof(THREADFAULTYPE), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(Goodsim, GoodSim, length*sizeof(int), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
	fault_detection_kernel<<<blocks,threads>>>(dev_table3,dev_res,Goodsim,length);


	HANDLE_ERROR( hipMemcpy(Final, dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));

    //for(i = 0; i<length; i++)printf("%d",GoodSim[i]);

    printf("length is %d\n",length);

    //for (i = 0; i<length; i++ )
    	//printf("%d",Final[i]);
}



extern "C" void device_allocations()
{
	size_t size = patterns*maxgates;
	//int dev;

	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);
	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
}


extern "C" void device_allocations2()
{
	size_t size = 100000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
}


extern "C" void alloc(){
	size_t size = 100000000;
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADTYPE)));
}


extern "C" void device_allocations3()
{
	int length = detect_index*patterns;
	//int length =100000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table3, length*sizeof(THREADFAULTYPE)));
	HANDLE_ERROR( hipMalloc( (void**)&Goodsim, length*sizeof(int)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, length*sizeof(int)));
}


extern "C" void device_deallocations()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());

}


extern "C" void dealloc(){
	HANDLE_ERROR( hipFree(dev_table2));
}


extern "C" void device_deallocations2()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table2));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());
}

extern "C" void device_deallocations3()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table3));
    HANDLE_ERROR( hipFree(dev_res));
    HANDLE_ERROR( hipDeviceReset());
}
*/
