#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADPTR dev_table = NULL;
THREADFAULTPTR dev_table2 = NULL;
RESULTPTR dev_res = NULL;
int *dev_LUT = NULL;
//int total=0;

__global__ void logic_simulation_kernel(THREADPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//elegxos an perasoume ta osa steilame
	//xreiazomaste kai allon ena elegxo an ayta pou exoume steilei einai pio polla apo ta nhmata
	if (tid < length) {
		THREADTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid].output = output;
	}
}


__global__ void fault_injection_kernel(THREADFAULTPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADFAULTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid].output = (output & data.m0)||data.m1;
	}
}


extern "C" void dummy_gpu(int level){
	//int i;
	int blocks;
	int threads;


	//size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[level];
	//total=total+length;
	//printf("Length is %d\n",total);

	//device_allocations();

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, cuda_tables[level], length*sizeof(THREADTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
    logic_simulation_kernel<<<blocks,threads>>>(dev_table,dev_res,length);


	HANDLE_ERROR( hipMemcpy(result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",result_tables[level][i]);
    //printf("\n");
}


extern "C" void dummy_gpu2(int level){
	int i;
	int blocks;
	int threads;

	//theloume ta no_po_faults na antistixoizontai me kapoio tropo me ta
	//faults tou kathe epipedou
	int length = no_po_faults*patterns;
	//total=total+length;
	//printf("Length is %d\n",total);

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, fault_tables, length*sizeof(THREADFAULTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
	fault_injection_kernel<<<blocks,threads>>>(dev_table2,dev_res,length);


	HANDLE_ERROR( hipMemcpy(fault_result_tables, dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",fault_result_tables[i]);
}




extern "C" void device_allocations()
{
	size_t size = patterns*maxgates;
	//int dev;

	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);
	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
}


extern "C" void device_allocations2()
{
	size_t size = 1000000;

	//HANDLE_ERROR( hipSetDevice (2));

    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table2, size*sizeof(THREADFAULTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
}




extern "C" void device_deallocations()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table));
    HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());

}

extern "C" void device_deallocations2()
{
    // Free device global memory
    HANDLE_ERROR( hipFree(dev_table2));
    HANDLE_ERROR( hipFree(dev_res));
    HANDLE_ERROR( hipDeviceReset());

}
