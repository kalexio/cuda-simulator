#include "hip/hip_runtime.h"
#include "define.h"
#include "structs.h"

extern "C" {
#include "my_cuda.h"
}

texture<int> texLUT;

THREADPTR dev_table = NULL;
RESULTPTR dev_res = NULL;
int *dev_LUT = NULL;

__global__ void logic_simulation_kernel(THREADPTR dev_table,RESULTPTR dev_res,int length){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < length) {
		THREADTYPE data = dev_table[tid];
		int index = data.offset + data.input[0] + data.input[1]*2 + data.input[2]*4 + data.input[3]*8;
		int output = tex1Dfetch(texLUT,index);
		dev_res[tid].output = output;
	}
}


extern "C" void dummy_gpu(int level){
	//int i;
	int blocks;
	int threads;

	//size_t size = patterns*levels[0]*sizeof(THREADTYPE);
	int length = patterns*levels[level];
	//printf("Length is %d\n",length);

	//device_allocations();

	//copy from Ram to device
	HANDLE_ERROR( hipMemcpy(dev_table, cuda_tables[level], length*sizeof(THREADTYPE), hipMemcpyHostToDevice));

	//printf("length of array=%d\n",length);
	//printf("maxgates=%d\n",maxgates);

	threads = 128;
	blocks = (length+(threads-1))/threads;
	if (blocks < 200) {
		threads = 64;
		blocks = (length+(threads-1))/threads;
	}
   // printf("The number of blocks %d\n",blocks);
    logic_simulation_kernel<<<blocks,threads>>>(dev_table,dev_res,length);


	HANDLE_ERROR( hipMemcpy(result_tables[level], dev_res,length*sizeof(int) , hipMemcpyDeviceToHost));


    //for (i = 0; i<length; i++ )
    	//printf("%d",result_tables[level][i]);
    //printf("\n");

    // Free device global memory
    //HANDLE_ERROR( hipFree(dev_table));
    //HANDLE_ERROR( hipFree(dev_res));
    //HANDLE_ERROR( hipDeviceReset());
}



extern "C" void device_allocations()
{
	size_t size = patterns*maxgates;
	int dev;

	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);
	HANDLE_ERROR( hipSetDevice (2));
	//HANDLE_ERROR( hipGetDevice (&dev));
	//printf("ID of current CUDA device: %d\n",dev);

	//allocations for texture memory
	HANDLE_ERROR( hipMalloc( (void**)&dev_LUT, 182*sizeof(int)));
    //allocations cuda table
	HANDLE_ERROR( hipMalloc( (void**)&dev_table, size*sizeof(THREADTYPE)));
	//allocations for result table
	HANDLE_ERROR( hipMalloc( (void**)&dev_res, size*sizeof(int)));
	//fill and bind the texture
	HANDLE_ERROR( hipMemcpy(dev_LUT, LUT, 182*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipBindTexture( NULL,texLUT,dev_LUT, 182*sizeof(int)));
}
